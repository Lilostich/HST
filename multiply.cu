#include "hip/hip_runtime.h"
//
// Created by Dan on 23.12.2022.
//
// #include <iostream>

// #include <vector>
// #include <math.h>
// #include <time.h>
#include <stdio.h>
// #include <fcntl.h>
// #include <stdlib.h>

///
/// \brief Умножение вектора на вектор
/// \param[in] lv - левый вектор
/// \param[in] vect - правый вектор
/// \param[in] size - размер векторов
/// \param[out] result - результат умножения
/// \param[out] debug - вектор для дебаг значений


// __global__ void kernelMultiply(double *lv, double *vect, double *result, int size, double* debug){
__global__ void kernelMultiply(double *lv, double *vect, double *result,int size, double* debug){
    // result[0] = 404.0;
    // lv[0] = 404.0;
    // vect[0] = 404.0;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
        *result += lv[i] * vect[i];
    // if (debug[0] < (double)(int)threadIdx.x)
        // debug[0] = threadIdx.x;
    // if (debug[1] < (double)(int)blockIdx.x)
        // debug[1] = blockIdx.x;
    // if (debug[2] < (double)(int)blockDim.x)
        // debug[2] = blockDim.x;
}



///
/// \brief Умножение вектора на вектор
/// \param[in] matrix - матрица
/// \param[in] vect - вектор
/// \param[out] value - результат умножения
/// \param size - размер векторов
///
double multiply_str_str(double *leftVector, double *rightVector,int size, time_t *time) {
    
    hipError_t error;
    int step = 0;
    double *d_rVector = NULL, *d_lVector = NULL, *d_result = NULL;
    double *result = (double *)malloc(sizeof(double));
    result[0] = -1;

    error = hipMalloc((void**)&d_rVector,sizeof(double) * size);
    // printf("step %d error is %d\n",step++,error);
    error = hipMalloc((void**)&d_lVector,sizeof(double) * size);
    // printf("step %d error is %d\n",step++,error);
    error = hipMalloc((void**)&d_result,sizeof(double) * 1);
    // printf("step %d error is %d\n",step++,error);

    // printf("leftVector is [%lf, %lf, ...]\n",leftVector[0],leftVector[1]);
    // printf("rightVector is [%lf, %lf, ...]\n",rightVector[0],rightVector[1]);

    error = hipMemcpy(d_rVector,rightVector,sizeof(double) * size, hipMemcpyHostToDevice);
    // printf("step %d error is %d\n",step++,error);
    error = hipMemcpy(d_lVector,leftVector ,sizeof(double) * size, hipMemcpyHostToDevice);
    // printf("step %d error is %d\n",step++,error);
    error = hipMemcpy(d_result,result ,sizeof(double), hipMemcpyHostToDevice);
    // printf("step %d error is %d\n",step++,error);

    // DEBUG
    int debug_size = 10;
    double* debug = (double*)malloc(sizeof(double) * debug_size);
    debug[0] = debug[1] = debug[2] = -1;
    double *d_debug;
    error = hipMalloc((void**)&d_debug,sizeof(double) * debug_size);
    // printf("step %d error is %d\n",step++,error);
    // \DEBUG

    error = hipMemcpy(d_debug,debug,sizeof(double) * debug_size, hipMemcpyHostToDevice);
    // printf("step %d error is %d\n",step++,error);
    time_t first = clock();
 
    // TRY RUN
    // kernelMultiply <<< 256, 256 >>> (d_lVector, d_rVector, d_result, size, d_debug);
    kernelMultiply <<< 1,1 >>> (d_lVector, d_rVector, d_result, size, d_debug);
    // printf("try run is is %d\n",error);
    hipDeviceSynchronize();
    hipDeviceSynchronize(); 
    time_t end = clock();

    error = hipMemcpy(debug,d_debug,sizeof(double) * debug_size, hipMemcpyDeviceToHost);
    // printf("step %d error is %d\n",step++,error);
    // printf("max threadIdx.x is %lf\n", debug[0]);
    // printf("max blockIdx.x is %lf\n", debug[1]);
    // printf("max blockDim.x is %lf\n", debug[2]);
    error = hipFree(d_debug);
    // printf("step %d error is %d\n",step++,error);
    // Вывод необходимых debug переменных
    free(debug);

    *time += end-first;
    
    error = hipMemcpyAsync(result, d_result, sizeof(double), hipMemcpyDeviceToHost);
    // printf("step %d error iss %d\n",step++,error);
    // printf("result is %lf\n",result[0]);
    

    error = hipFree(d_rVector);
    // printf("step %d error is %d\n",step++,error);
    error = hipFree(d_lVector);
    // printf("step %d error is %d\n",step++,error);
    error = hipFree(d_result);
    // printf("step %d error is %d\n",step++,error);
    // printf("All steps ends\n");

    return *result;
}



