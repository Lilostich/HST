#include "hip/hip_runtime.h"
//
// Created by Dan on 23.12.2022.
//
#include <iostream>

#include <vector>
#include <math.h>
#include <time.h>
#include <fstream>
#include <fcntl.h>

///
/// \brief Умножение вектора на вектор
/// \param[in] matrix - матрица
/// \param[in] vect - вектор
/// \param[out] value - результат умножения
/// \param size - размер векторов
///
__global__ void kernelMultiply(double *lv, double *vect, double *result,double* debug, int size,int line){
    int i = threadIdx.x;
//    int i = blockIdx.x;
    result[line] += lv[i] * vect[i];

    debug[0] = threadIdx.x;
    debug[1] = blockIdx.x;
    debug[2] = blockDim.x;
    debug[3] = 5.6;
    debug[5] = tid;
    debug[6] = matrix[tid * size + 0];
    debug[7] = vect[0];
    debug[8] = size;
    debug[9] = matrix[6];

}

double multiply_str_str(std::vector<double> &leftVector, std::vector<double> &rightVector,time_t *time) {
    std::vector<double> _values(leftVector);
    std::vector<double> res;

    int size = rightVector.size();
    double *rVector = (double*)malloc(sizeof(double) * size);
    double *matrix = (double*)malloc(sizeof(double) * size);
    double *result = (double*)malloc(sizeof(double) * size);

    double *d_rVector, *d_matrix, *d_result;

    for(int i = 0; i < size; i++){
        rVector[i] = rightVector[i];
        result[i] = 0;
    }
    for(int i = 0; i < size * size; i++){
        matrix[i] = _values[i];
    }

    hipMalloc((void**)&d_rVector,sizeof(double) * size);
    hipMalloc((void**)&d_matrix,sizeof(double) * size * size);
    hipMalloc((void**)&d_result,sizeof(double) * size);

    hipMemcpy(d_rVector,rVector,sizeof(double) * size,      hipMemcpyHostToDevice);
    hipMemcpy(d_matrix,matrix,sizeof(double) * size * size, hipMemcpyHostToDevice);

    double* debug = (double*)malloc(sizeof(double) * 10);
    double *d_debug;
    hipMalloc((void**)&d_debug,sizeof(double) * 10);
    hipMemcpy(d_debug,debug,sizeof(double) * 10, hipMemcpyHostToDevice);
    time_t first = clock();
    kernelMultiply <<<size, size >>> (d_matrix, d_rVector, d_result, d_debug, size);
    time_t end = clock();

    hipMemcpy(debug,d_debug,sizeof(double) * 10, hipMemcpyDeviceToHost);
    hipFree(d_debug);
    free(debug);

    *time += end-first;

    hipMemcpy(result, d_result, sizeof(double) * size, hipMemcpyDeviceToHost);

    hipFree(d_rVector);
    hipFree(d_matrix);
    hipFree(d_result);

    free(rVector);
    free(matrix);

    res.reserve(size);
    res.resize(size);
    for(int i = 0; i < size; i++){
        res[i] = result[i];
    }

    free(result);
    return res;
}



