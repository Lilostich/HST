
#include <hip/hip_runtime.h>
#include <iostream>

#include <vector>
#include <math.h>
#include <time.h>
#include <fstream>
#include <fcntl.h>

class Matrix {
public:
    Matrix(std::vector<double> matr);
    Matrix(std::string fileName);
    Matrix(){}
    void initSize(int n);

    double& getValue(int i, int j);

    std::vector<double> multiply(std::vector<double> rightVector,time_t *sum);

    void save(std::string fileName);
    int getSize(){return size;}
private:
    std::vector<double> values;
    int size;
};


Matrix::Matrix(std::string fileName) {
    FILE *fd;

    fd = fopen(fileName.c_str(),"r+b");
    fscanf(fd,"%d ",&size);
    double cell = 0;
    for (int i = 0; i < size; i++){
        for (int j = 0; j < size; j++){
            fscanf(fd,"%lf ",&cell);
            values.push_back(cell);
        }
    }

    fclose(fd);
}

///
/// \brief Умножение вектора на вектор
/// \param[in] matrix - матрица
/// \param[in] vect - вектор
/// \param[out] value - результат умножения
/// \param size - размер векторов
///
__global__ void kernelMultiply(double *matrix, double *vect, double *result,double* debug, int size){
    int tid = (threadIdx.x + blockIdx.x * blockDim.x)%size;
    int i = threadIdx.x;
    int j = blockIdx.x;
    result[i] += matrix[i * size + j] * vect[j];

    debug[0] = threadIdx.x;
    debug[1] = blockIdx.x;
    debug[2] = blockDim.x;
    debug[3] = 5.6;
    debug[5] = tid;
    debug[6] = matrix[tid * size + 0];
    debug[7] = vect[0];
    debug[8] = size;
    debug[9] = matrix[6];

}

Matrix::Matrix(std::vector<double> matr) :values(matr){size = sqrt(matr.size());}

void Matrix::initSize(int n) {values.reserve(n*n); size = n*n;}

double &Matrix::getValue(int i, int j) {return values[j + i * size];}

std::vector<double> Matrix::multiply(std::vector<double> rightVector,time_t *time) {
    std::vector<double> _values(values);
    std::vector<double> res;
//    printf("1\n");

    double *rVector = (double*)malloc(sizeof(double) * size);
    double *matrix = (double*)malloc(sizeof(double) * size * size);
    double *result = (double*)malloc(sizeof(double) * size);

    double *d_rVector, *d_matrix, *d_result;

//    for(int ii = 0; ii < res.size(); ii++){
//    }


    for(int i = 0; i < size; i++){
        rVector[i] = rightVector[i];
        result[i] = 0;
//        if(i < 10)
//            std::cout << rVector[i] << " ";
    }
//    std::cout << std::endl;
//    printf("\n");
    for(int i = 0; i < size * size; i++){
        matrix[i] = _values[i];
//        if(i < 10)
//            std::cout << matrix[i] << " ";
    }
//    std::cout << std::endl;

    hipMalloc((void**)&d_rVector,sizeof(double) * size);
    hipMalloc((void**)&d_matrix,sizeof(double) * size * size);
    hipMalloc((void**)&d_result,sizeof(double) * size);
//    printf("asdasd2\n");

    hipMemcpy(d_rVector,rVector,sizeof(double) * size,      hipMemcpyHostToDevice);
    hipMemcpy(d_matrix,matrix,sizeof(double) * size * size, hipMemcpyHostToDevice);
//    printf("asdasd3\n");

//    cudaEvent_t start,stop;
    float gpuTime;
//    cudaEventCreate(&start);
//    cudaEventCreate(&stop);
//    cudaEventRecord(start, 0);
//    printf("asdasd4\n");
    double* debug = (double*)malloc(sizeof(double) * 10);
    double *d_debug;
    hipMalloc((void**)&d_debug,sizeof(double) * 10);
    hipMemcpy(d_debug,debug,sizeof(double) * 10, hipMemcpyHostToDevice);
    time_t first = clock();
    kernelMultiply <<<size, size >>> (d_matrix, d_rVector, d_result, d_debug, size);
    time_t end = clock();

    hipMemcpy(debug,d_debug,sizeof(double) * 10, hipMemcpyDeviceToHost);
//    for(int j = 0; j < 10; j++){
//        std::cout << "deb " << j <<" "  << debug[j] << std::endl;
//    }
    hipFree(d_debug);
    free(debug);

    *time += end-first;
//    printf("asdasd5\n");

//    cudaEventRecord(stop, 0);
//    cudaEventSynchronize(stop);
//    cudaEventElapsedTime(&gpuTime, start, stop);
//    printf("time on GPU = %.2lf ms \n", gpuTime);
//    cudaEventDestroy(start);
//    cudaEventDestroy(stop);
//    printf("asdasd6\n");

    hipMemcpy(result, d_result, sizeof(double) * size, hipMemcpyDeviceToHost);
//    printf("asdasd7\n");

    hipFree(d_rVector);
    hipFree(d_matrix);
    hipFree(d_result);
//    printf("asdasd8\n");


    free(rVector);
    free(matrix);
//    printf("asdasd9\n");

    res.reserve(size);
    res.resize(size);
//    printf("asdasd10\n");
    for(int i = 0; i < size; i++){
        res[i] = result[i];
//        if(i < 10)
//            std::cout << result[i] << " " << res[i] << std::endl;
    }

//    for(int ii = 0; ii < res.size(); ii++){
//        if(ii < 10)
//            std::cout << result[ii] << " ";
//    }
//    std::cout << std::endl;

    free(result);
//    printf("asdasd11\n");
    return res;
}

void Matrix::save(std::string fileName) {
    std::ofstream fd;

    fd.open(fileName);
    fd << this->size << " ";
    for(int i = 0; i < size; i++){
        for (int j = 0; j < size; j++){
            fd << values[i * size + j] << " ";
        }
    }
}

const double ACCURACY = 0.00001;
const std::string fileNames[] {"testMatrix.txt",
                               "matrixFile10.txt",
                               "matrixFile50.txt",
                               "matrixFile100.txt",
                               "matrixFile250.txt",
                               "matrixFile500.txt"};
const size_t FILE_COUNT = 6;
const size_t MAX_ITERATION = 100;

/// Длина вектора
double _dist(const std::vector<double> &rv){
    double res = 0;
    for (unsigned int i = 0; i < rv.size(); i++) {
        res += rv[i] * rv[i];
    }
    res = sqrt(res);
    return res != 0 ? res : 1;
}

/// деление вектора на значение
std::vector<double> _div(std::vector<double> rv, double diver){
    if (diver == 0)
        return rv;
    for (unsigned int i = 0; i < rv.size(); i++){
        rv[i] /= diver;
    }
    return rv;
}

void initRand(std::vector<double> &rv, int size){
//    printf("initRand\n");/

    rv.clear();
    rv.reserve(size);
    for (int i = 0; i < size; i++){
        rv.push_back(double(rand()));
    }
//    printf("endinitRand\n");

}

bool isVectorsNear(std::vector<double> &v1,std::vector<double> &v2){
//    printf("isVectorNear\n");
    for (int i = 0; i < v1.size(); i++){
        if (fabs(v1[i] - v2[i]) > ACCURACY)
        {
//            printf("endisVectorNear\n");
            return false;
        }
    }
//    printf("endisVectorNear\n");
    return true;
}

void findMyValue(Matrix &m,std::vector<double> &rv){
//    printf("findMyValue\n");
    std::vector<double> rvold;
    time_t sum = 0;
    int j = 0;
//    printf("%d\n",j++);
    time_t a = clock();
    time_t summ = 0;
    for (int i = 0; i < MAX_ITERATION; i++){
        rvold = rv;
//        printf("fmv1 %d\n",i);
//        for(int ii = 0; ii < rv.size(); ii++){
//            std::cout << rv[ii] << std::endl;
//        }
        std::vector<double> aa = m.multiply(rv,&summ);

        double distt = _dist(aa);
//        printf("fmv3 %d\n",i);
        rv = _div(aa, distt);
//        printf("fmv4 %d\n",i);
        if (isVectorsNear(rv,rvold))
            break;
//        printf("fmv5 %d\n",i);
    }
//    printf("%d\n",j++);

    time_t b = clock();
    sum += b - a;
    std::cout << "time is " << sum << std::endl;
    std::cout << "real time is " << summ << std::endl;
//    printf("endfindMyValue\n");
}

void saveVector(std::vector<double> &myVector, std::string fileName) {
//    printf("saveVector\n");
    FILE *fd;
    fd = fopen(("result_" + fileName).c_str(), "w+t");
    for (const auto &value : myVector){
        fprintf(fd, "%lf ", value);
    }
    fclose(fd);
//    printf("endSaveVector\n");
}

int main(int argc, char *argv[])
{
    for (int i = 0; i < FILE_COUNT; i++){
//        printf("iteration %d\n",i);
        Matrix matrix(fileNames[i]);
        int j = 0;
//        printf("%d\n",j++);
        std::vector<double> myVector;
        initRand(myVector,matrix.getSize());
//        printf("%d\n",j++);
        findMyValue(matrix,myVector);
//        printf("%d\n",j++);
        saveVector(myVector,fileNames[i]);
//        printf("%d\n",j++);
//        printf("enditeration %d\n",i);
    }
    return EXIT_SUCCESS;
}

/*
time is 166
time is 23
time is 111
time is 246
time is 688
time is 1274
 */